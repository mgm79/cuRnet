#include "hip/hip_runtime.h"
#define THRUST_IGNORE_CUB_VERSION_CHECK

#include "cudaGraph.cuh"

#include "../../../timer.h"

namespace scc4k{

	__device__ int duplicateCounter;
	__device__ int devF2Size[4];


	cudaGraph::cudaGraph(curnet::base_graph& _graph) : graph(_graph){
		hipError_t("At constructor!");
		V = graph.v();
		E = graph.e();
	
	TIMEHANDLE start = start_time();
	//#define USE_SINGLE_CUDAMALLOC
	#ifdef USE_SINGLE_CUDAMALLOC
	  //globalPointer
	  size_t globalSize = 
	  (V + 1) * sizeof (int) +
		E * sizeof (int) +
		(V + 1) * sizeof (int) +
		E * sizeof (int) +
		V * sizeof (dist_t) +
		V*2 * sizeof (color_t) +
		V * sizeof (uint8_t) +
		(V*3+1) * sizeof (int) +
		(V*3+1) * sizeof (int) +
		V * sizeof(dist_t) +
		sizeof(int) +
		sizeof(int) +
		sizeof(int);
	
		// --------------- Frontier Allocation -------------------
		hipError_t("Before call meminfo");
		size_t free, total;
		hipMemGetInfo(&free, &total);
	
		hipError_t("After call meminfo");
		size_t frontierSize = ((free - globalSize) / 8) - 2* 500 * 1024;
	 
		//std::cout << "Frontier size: " <<  frontierSize;
		//frontierSize = min(static_cast<int>((1<<27 - globalSize) >> 1), static_cast<int>(frontierSize)); //empirical limit value before crushing malloc performance!
		//std::cout << " - " <<  frontierSize << std::endl;
	 
		globalSize += frontierSize + frontierSize;
	
		//std::cout << globalSize << std::endl;

		allocFrontierSize = frontierSize / sizeof(int);

		hipError_t("Graph Frontier Allocation");
	
		size_t gS[15] = { (V + 1) * sizeof (int), E * sizeof (int), (V + 1) * sizeof (int), E * sizeof (int), V * sizeof (dist_t), V*2 * sizeof (color_t), V * sizeof (uint8_t), 
						(V*3+1) * sizeof (int),	(V*3+1) * sizeof (int),	V * sizeof(dist_t), sizeof(int), sizeof(int), sizeof(int), frontierSize, frontierSize};
		size_t gSC[15];
		gSC[0] = 0;
		for(int i = 1; i < 15; ++i) { gSC[i] = gSC[i-1] + gS[i-1]; }
	 
		hipError_t("before hipMemcpy");
		hipMalloc((void **) &globalPointer, globalSize);
		hipError_t("hipMemcpy global");
		devOutNodes = (int*) (globalPointer + gSC[0]);
		devOutEdges = (int*) (globalPointer + gSC[1]);
		devInNodes = (int*) (globalPointer + gSC[2]);
		devInEdges = (int*) (globalPointer  + gSC[3]);

		devDistance = (dist_t*) (globalPointer + gSC[4]);
		devColor = (int*) (globalPointer + gSC[5]);
		devState = (uint8_t*) (globalPointer + gSC[6]);
		devPivotPerColor = (int*) (globalPointer + gSC[7]);
		devPivotPerColorSize = (int*) (globalPointer +  + gSC[8]);
		devDistanceIn = (dist_t*) (globalPointer + gSC[9]);
		devGlobalCounter = (int*) (globalPointer + gSC[10]);
	
		D_hasToContinue = (int*) (globalPointer + gSC[11]);
		D_num_of_scc = (int*) (globalPointer + gSC[12]);
	
		devF1 = (int*)  (globalPointer + gSC[13]);
		devF2 = (int*)  (globalPointer + gSC[14]);
	#else
		hipError_t("Before all malloc");
		hipMalloc((void **) &devOutNodes, (V + 1) * sizeof (int));
		hipMalloc((void **) &devOutEdges, E * sizeof (int));
		hipMalloc((void **) &devInNodes, (V + 1) * sizeof (int));
		hipMalloc((void **) &devInEdges, E * sizeof (int));
		//hipMalloc(&devF1, V * (F_MUL) * sizeof (int));
		//hipMalloc(&devF2, V * (F_MUL) * sizeof (int));
		hipMalloc((void **) &devDistance, V * sizeof (dist_t));

		hipMalloc((void **) &devColor, V*2 * sizeof (color_t));
		hipMalloc((void **) &devState, V * sizeof (uint8_t));
		hipMalloc((void **) &devPivotPerColor, (V*3+1) * sizeof (int));
		hipMalloc((void **) &devPivotPerColorSize, (V*3+1) * sizeof (int));
		hipMalloc((void **) &devDistanceIn, V * sizeof(dist_t));

		hipMalloc((void **) &devGlobalCounter, sizeof(int));
	
		hipMalloc((void **) &D_hasToContinue, sizeof(int));
		hipMalloc((void **) &D_num_of_scc, sizeof(int));
		hipError_t("After all malloc");
	
		// --------------- Frontier Allocation -------------------

		size_t free, total;
		hipMemGetInfo(&free, &total);
		size_t frontierSize = (free / 32) - 2* 500 * 1024;
		//frontierSize = min( frontierSize, static_cast<size_t>(1 << 10*10*7) ); //max 128 MB allocation
	 
		//std::cout << "Frontier size: " <<  frontierSize;
		//frontierSize = min(frontierSize, static_cast<size_t>(1 << 27));
		//std::cout << " - " <<  frontierSize << std::endl;
		hipMalloc((void **) &devF1, frontierSize);
		hipMalloc((void **) &devF2, frontierSize);

		allocFrontierSize = frontierSize / sizeof(int);

		hipError_t("Graph Frontier Allocation");
	 #endif
		hipMemcpy((void**) devOutNodes, graph.v_out(), (V + 1) * sizeof (int), hipMemcpyHostToDevice);
		hipMemcpy((void**) devOutEdges, graph.e_out(), E * sizeof (int), hipMemcpyHostToDevice);
		hipError_t("hipMemcpy");

		double ptime = end_time(start);
//		std::cout << "Construction: " << ptime << std::endl;	

		hipMemcpy((void**) devInNodes, graph.v_in(), (V + 1) * sizeof (int), hipMemcpyHostToDevice);
		hipMemcpy((void**) devInEdges, graph.e_in(), E * sizeof (int), hipMemcpyHostToDevice);

		frontier = new int[V];

		const int ZERO = 0;
		hipMemcpyToSymbol(HIP_SYMBOL(duplicateCounter), &ZERO, sizeof (int));

		hipError_t("Graph Allocation");

	}


	cudaGraph::~cudaGraph()
	{
		hipError_t("Init Graph Deallocation");
		TIMEHANDLE t1 = start_time();	
		delete[] frontier;
	#ifdef USE_SINGLE_CUDAMALLOC
		hipFree(globalPointer);
		hipError_t("free global pointer");
	#else
		hipError_t("Init Graph Deallocation");
		hipFree(devOutNodes);
		hipFree(devOutEdges);
		hipFree(devInNodes);
		hipFree(devInEdges);
		hipFree(devDistance);
		hipError_t("Mid");
		hipFree(devColor);
		hipFree(devState);
		hipFree(devPivotPerColor);
		hipFree(devPivotPerColorSize);
		hipFree(devDistanceIn);

		hipFree(devGlobalCounter);

		hipFree(D_hasToContinue);
		hipFree(D_num_of_scc);

		hipFree(devF1);
		hipFree(devF2);
		hipError_t("Graph Deallocation");
	#endif
		double ptime = end_time(t1);
//		std::cout << "Destuction: " << ptime << std::endl;
	}

	bool cudaGraph::copyColorToVector(std::vector<int> &c)
	{
		int *colors = new int[V*2]();
		hipMemcpy(colors, (void*) (devColor), (2*V) * sizeof (color_t), hipMemcpyDeviceToHost);
		c.clear();
		for(int i = 0; i < V*2; i+=2) c.push_back(-colors[i]);

		delete[] colors;
		return true;
	}
	bool cudaGraph::copyDistanceToVector(dist_t* v)
	{
		hipError_t("Init copy distance");
		hipMemcpy(v, (void*) (devDistance), (V) * sizeof (dist_t), hipMemcpyDeviceToHost);
		hipError_t("End copy distance");
		return true;
	}
	bool cudaGraph::copyDistanceToVector(std::vector<float> &v)
	{
		hipError_t("Init copy distance");
		dist_t* vv = new dist_t[V];
		hipMemcpy(vv, (void*) (devDistance), (V) * sizeof (dist_t), hipMemcpyDeviceToHost);
		for(int i = 0; i < V; i++)
		{
			//if(i < 100) std::cout << vv[i] << ", " << INF << std::endl;
			
			if(vv[i] == INF)
				v[i] = std::numeric_limits<float>::infinity();
			else
			{
				//std::cout << "No inf!" << std::endl;
				v[i] = static_cast<float>(vv[i]);
				//std::cout << v[i] << std::endl;
			}
			//v[i] = static_cast<float>(vv[i]);
				
		}
		delete[] vv;
		hipError_t("End copy distance");
		return true;
	}
}

#include "WorkEfficientKernel/BFS_WE_Kernels1.cu"
#include "WorkEfficientKernel/BFS_WE_Dynamic.cu"

// ----------------------- GLOBAL SYNCHRONIZATION --------------------------------

#define __GLOBAL_DEVICE__ __global__
#define NAME1 BFS_KernelMainGLOB
#define NAME1B BFS_KernelMainGLOBB

#include "WorkEfficientKernel/BFS_WE_KernelMain.cu"

#undef __GLOBAL_DEVICE__
#undef NAME1
#undef NAME1B

#define __GLOBAL_DEVICE__ __device__
#define NAME1 BFS_KernelMainDEV
#define NAME1B BFS_KernelMainDEVB

#include "WorkEfficientKernel/BFS_WE_KernelMain.cu"

#undef __GLOBAL_DEVICE__
#undef NAME1
#undef NAME1B

// ----------------------------------------------------------------------------------

//#include "WorkEfficientKernel/BFS_WE_KernelDispatch.cu"
#include "Util/GlobalSync.cu"
//#include "WorkEfficientKernel/BFS_WE_Block.cu"

#include "BFS_WorkEfficient.cu"
